#include<hip/hip_runtime.h>
#include<stdio.h>

// this function will run on gpu
__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}

int main() {
    // <<<M,T> M is thread blocks and T is parallel threads in each thread block
    // so cuda_hello will run 4 times
    cuda_hello<<<2,2>>>(); 
    return 0;
}